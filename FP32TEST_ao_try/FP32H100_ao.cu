#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <iomanip>
#include <vector>
#include <string>
#include <cctype>
#include <cmath>
#include <map>
#include <cstdint>

// 操作码枚举
enum Opcode {
    ADD, SUB, MUL, FMA, FMS, FNMA, FNMS,
    CMPEQ, CMPLT, CMPLE, CMPGT,
    CMPLTNUM, CMPLENUM, CMPGTNUM, UNORDERED
};

// 舍入模式枚举
enum RoundMode {
    RND_ZERO, RND_MINUS_INF, RND_PLUS_INF, RND_NEAREST
};

// 测试用例结构
struct TestCase {
    Opcode opcode;
    RoundMode roundMode;
    uint32_t operandA;
    uint32_t operandB;
    uint32_t operandC;
};

// 结果结构
struct Result {
    uint32_t result;
};

// 字符串到操作码映射
std::map<std::string, Opcode> opcodeMap = {
    {"ADD", ADD}, {"SUB", SUB}, {"MUL", MUL}, {"FMA", FMA}, {"FMS", FMS},
    {"FNMA", FNMA}, {"FNMS", FNMS}, {"CMPEQ", CMPEQ}, {"CMPLT", CMPLT},
    {"CMPLE", CMPLE}, {"CMPGT", CMPGT}, {"CMPLTNUM", CMPLTNUM},
    {"CMPLENUM", CMPLENUM}, {"CMPGTNUM", CMPGTNUM}, {"UNORDERED", UNORDERED}
};

// 字符串到舍入模式映射
std::map<std::string, RoundMode> roundModeMap = {
    {"RND_ZERO", RND_ZERO}, {"RND_MINUS_INF", RND_MINUS_INF},
    {"RND_PLUS_INF", RND_PLUS_INF}, {"RND_NEAREST", RND_NEAREST}
};

// CUDA内核：执行测试用例
__global__ void executeTests(const TestCase* __restrict__ testCases, 
                             Result* __restrict__ results, 
                             int numTests) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= numTests) return;
    
    // 使用寄存器存储减少内存访问
    TestCase tc = testCases[idx];
    const float a = __uint_as_float(tc.operandA);
    const float b = __uint_as_float(tc.operandB);
    const float c = __uint_as_float(tc.operandC);
    float res = 0.0f;
    
    // 分支预测优化
    switch (tc.opcode) {
        case ADD:
            switch (tc.roundMode) {
                case RND_ZERO: res = __fadd_rz(a, c); break;
                case RND_MINUS_INF: res = __fadd_rd(a, c); break;
                case RND_PLUS_INF: res = __fadd_ru(a, c); break;
                default: res = __fadd_rn(a, c);
            }
            break;
        case SUB:
            switch (tc.roundMode) {
                case RND_ZERO: res = __fsub_rz(a, c); break;
                case RND_MINUS_INF: res = __fsub_rd(a, c); break;
                case RND_PLUS_INF: res = __fsub_ru(a, c); break;
                default: res = __fsub_rn(a, c);
            }
            break;
        case MUL:
            switch (tc.roundMode) {
                    case RND_ZERO: res = __fmul_rz(a, b); break;
                    case RND_MINUS_INF: res = __fmul_rd(a, b); break;
                    case RND_PLUS_INF: res = __fmul_ru(a, b); break;
                    default: res = __fmul_rn(a, b);
            }
            break;
        case FMA:
            switch (tc.roundMode) {
                case RND_ZERO: res = __fmaf_rz(a, b, c); break;
                case RND_MINUS_INF: res = __fmaf_rd(a, b, c); break;
                case RND_PLUS_INF: res = __fmaf_ru(a, b, c); break;
                default: res = __fmaf_rn(a, b, c);
            }
            break;
        case FMS:
            switch (tc.roundMode) {
                case RND_ZERO: res = __fmaf_rz(a, b, -c); break;
                case RND_MINUS_INF: res = __fmaf_rd(a, b, -c); break;
                case RND_PLUS_INF: res = __fmaf_ru(a, b, -c); break;
                default: res = __fmaf_rn(a, b, -c);
            }
            break;
        case FNMA:
            switch (tc.roundMode) {
                case RND_ZERO: res = __fmaf_rz(-a, b, c); break;
                case RND_MINUS_INF: res = __fmaf_rd(-a, b, c); break;
                case RND_PLUS_INF: res = __fmaf_ru(-a, b, c); break;
                default: res = __fmaf_rn(-a, b, c);
            }
            break;
        case FNMS:
            switch (tc.roundMode) {
                case RND_ZERO: res = __fmaf_rz(-a, b, -c); break;
                case RND_MINUS_INF: res = __fmaf_rd(-a, b, -c); break;
                case RND_PLUS_INF: res = __fmaf_ru(-a, b, -c); break;
                default: res = __fmaf_rn(-a, b, -c);
            }
            break;
        case CMPEQ:
            res = (a == c) ? __int_as_float(0xFFFFFFFF) : 0.0f;
            break;
        case CMPLT:
            res = (a < c) ? __int_as_float(0xFFFFFFFF) : 0.0f;
            break;
        case CMPLE:
            res = (a <= c) ? __int_as_float(0xFFFFFFFF) : 0.0f;
            break;
        case CMPGT:
            res = (a > c) ? __int_as_float(0xFFFFFFFF) : 0.0f;
            break;
        case CMPLTNUM:
            res = fminf(a, c);
            break;
        case CMPLENUM:
            res = fminf(a, c);
            break;
        case CMPGTNUM:
            res = fmaxf(a, c);
            break;
        case UNORDERED:
            res = (isnan(a) || isnan(c)) ? __int_as_float(0xFFFFFFFF) : 0.0f;
            break;
    }
    
    // 存储结果
    results[idx].result = __float_as_uint(res);
}

// 解析十六进制字符串
uint32_t parseHex(const std::string& hexStr) {
    return std::stoul(hexStr, nullptr, 16);
}

// 读取输入文件
std::vector<TestCase> readInputFile(const std::string& filename) {
    std::vector<TestCase> testCases;
    std::ifstream file(filename);
    
    // 检查文件是否成功打开
    if (!file.is_open()) {
        std::cerr << "错误：无法打开输入文件 " << filename << std::endl;
        return testCases;
    }
    
    std::string line;
    while (std::getline(file, line)) {
        std::istringstream iss(line);
        std::string token;
        std::vector<std::string> tokens;
        
        while (std::getline(iss, token, ',')) {
            token.erase(0, token.find_first_not_of(' '));
            token.erase(token.find_last_not_of(' ') + 1);
            tokens.push_back(token);
        }
        
        if (tokens.size() == 5) {
            TestCase tc;
            if (opcodeMap.find(tokens[0]) != opcodeMap.end()) {
                tc.opcode = opcodeMap[tokens[0]];
            } else {
                std::cerr << "未知操作码: " << tokens[0] << std::endl;
                continue;
            }
            
            if (roundModeMap.find(tokens[1]) != roundModeMap.end()) {
                tc.roundMode = roundModeMap[tokens[1]];
            } else {
                std::cerr << "未知舍入模式: " << tokens[1] << std::endl;
                continue;
            }
            
            tc.operandA = parseHex(tokens[2]);
            tc.operandB = parseHex(tokens[3]);
            tc.operandC = parseHex(tokens[4]);
            testCases.push_back(tc);
        }
    }
    
    if (testCases.empty()) {
        std::cerr << "警告：输入文件中没有找到有效测试用例\n";
    }
    
    return testCases;
}

// 写输出文件
void writeOutputFile(const std::string& filename, 
                    const std::vector<TestCase>& testCases,
                    const std::vector<Result>& results) {
    std::ofstream file(filename);
    if (!file.is_open()) {
        std::cerr << "错误：无法创建输出文件 " << filename << std::endl;
        return;
    }
    
    file << "Opcode, Rnd, Operand A, Operand B, Operand C, Result\n";
    
    // 反向映射用于输出
    std::map<Opcode, std::string> opcodeStr;
    for (const auto& p : opcodeMap) opcodeStr[p.second] = p.first;
    
    std::map<RoundMode, std::string> roundModeStr;
    for (const auto& p : roundModeMap) roundModeStr[p.second] = p.first;
    
    for (size_t i = 0; i < testCases.size(); ++i) {
        const TestCase& tc = testCases[i];
        const Result& res = results[i];
        
        file << opcodeStr[tc.opcode] << ", "
             << roundModeStr[tc.roundMode] << ", "
             << "0x" << std::hex << std::setw(8) << std::setfill('0') << tc.operandA << ", "
             << "0x" << std::hex << std::setw(8) << std::setfill('0') << tc.operandB << ", "
             << "0x" << std::hex << std::setw(8) << std::setfill('0') << tc.operandC << ", "
             << "0x" << std::hex << std::setw(8) << std::setfill('0') << res.result << "\n";
    }
}

int main(int argc, char* argv[]) {
    std::string inputFilename = "fp32_input.txt";
    std::string outputFilename = "h100_fp32_output.txt";
    
    // 从命令行获取文件名
    if (argc >= 2) {
        inputFilename = argv[1];
    }
    if (argc >= 3) {
        outputFilename = argv[2];
    }
    
    // 检查输入文件是否存在
    std::ifstream testFile(inputFilename);
    if (!testFile.good()) {
        std::cerr << "错误：文件 " << inputFilename << " 不存在，程序终止。\n";
        return 1;
    }
    testFile.close();
    
    // 读取输入文件
    std::vector<TestCase> testCases = readInputFile(inputFilename);
    if (testCases.empty()) {
        std::cerr << "错误：无有效测试用例，程序终止。\n";
        return 1;
    }
    
    int numTests = testCases.size();
    std::cout << "找到 " << numTests << " 个测试用例，开始处理...\n";
    
    // 分配设备内存
    TestCase* d_testCases;
    Result* d_results;
    hipMalloc(&d_testCases, numTests * sizeof(TestCase));
    hipMalloc(&d_results, numTests * sizeof(Result));
    
    // 创建CUDA流
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    // 异步拷贝数据到设备
    hipMemcpyAsync(d_testCases, testCases.data(), numTests * sizeof(TestCase), 
                   hipMemcpyHostToDevice, stream);
    
    // 启动内核
    int blockSize = 512;
    int gridSize = (numTests + blockSize - 1) / blockSize;
    executeTests<<<gridSize, blockSize, 0, stream>>>(d_testCases, d_results, numTests);
    
    // 异步拷贝结果回主机
    std::vector<Result> results(numTests);
    hipMemcpyAsync(results.data(), d_results, numTests * sizeof(Result), 
                   hipMemcpyDeviceToHost, stream);
    
    // 同步流
    hipStreamSynchronize(stream);
    
    // 写输出文件
    writeOutputFile(outputFilename, testCases, results);
    
    // 清理资源
    hipFree(d_testCases);
    hipFree(d_results);
    hipStreamDestroy(stream);
    
    std::cout << "H100 FP32 测试完成，结果已写入 " << outputFilename << std::endl;
    return 0;
}